#include "hip/hip_runtime.h"
#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>


__global__ void vectorAdd(const float *A, const float *B, float *C) {
    
    int i = blockDim.x*blockIdx.x + threadIdx.x;
    //for(int it = 0; it<10000; it++)
    C[i] = A[i] + B[i];
}



int main(int argc, char **argv){
    // Error code to check return values for CUDA calls
  hipError_t err = hipSuccess;

  // Print the vector length to be used, and compute its size
  int numElements = 1024;

  // Number of threads per block and number of blocks
  int threadsPerBlock = 1024;

  // printf("[Vector addition of %d elements]\n", numElements);
  if (checkCmdLineFlag(argc, (const char **)argv, "help") ||
      checkCmdLineFlag(argc, (const char **)argv, "?")) {
    printf("      -nN (Vector Size)\n");
    printf("      -nT (Number of Threads per Block)\n");
    exit(EXIT_SUCCESS);
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "nN")){
    numElements = getCmdLineArgumentInt(argc, (const char **)argv, "nN");
  }

  if (checkCmdLineFlag(argc, (const char **)argv, "nT")){
    threadsPerBlock = getCmdLineArgumentInt(argc, (const char **)argv, "nT");
  }

  size_t size = numElements * sizeof(float);
  int blocksPerGrid = numElements / threadsPerBlock;

  // Allocate the host input vector A
  float *h_A = (float *)malloc(size);

  // Allocate the host input vector B
  float *h_B = (float *)malloc(size);

  // Allocate the host output vector C
  float *h_C = (float *)malloc(size);

  // Verify that allocations succeeded
  if (h_A == NULL || h_B == NULL || h_C == NULL) {
    fprintf(stderr, "Failed to allocate host vectors!\n");
    exit(EXIT_FAILURE);
  }

  // Initialize the host input vectors
  for (int i = 0; i < numElements; ++i) {
    h_A[i] = rand() / (float)RAND_MAX;
    h_B[i] = rand() / (float)RAND_MAX;
  }

  // Allocate the device input vector A
  float *d_A = NULL;
  err = hipMalloc((void **)&d_A, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Allocate the device input vector B
  float *d_B = NULL;
  err = hipMalloc((void **)&d_B, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Allocate the device output vector C
  float *d_C = NULL;
  err = hipMalloc((void **)&d_C, size);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy the host input vectors A and B in host memory to the device input
  // vectors in
  // device memory
  // printf("Copy input data from the host memory to the CUDA device\n");
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector A from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector B from host to device (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  //int threadsPerBlock = 256;
  //int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
  // printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid,
  //        threadsPerBlock);
  vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C);
  err = hipGetLastError();

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Copy the device result vector in device memory to the host result vector
  // in host memory.
  //printf("Copy output data from the CUDA device to the host memory\n");
  err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

  if (err != hipSuccess) {
    fprintf(stderr,
            "Failed to copy vector C from device to host (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Verify that the result vector is correct
  for (int i = 0; i < numElements; ++i) {
    if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5) {
      fprintf(stderr, "Result verification failed at element %d!\n", i);
      exit(EXIT_FAILURE);
    }
  }

  printf("Test PASSED\n");

  // Free device global memory
  err = hipFree(d_A);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector A (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_B);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector B (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  err = hipFree(d_C);

  if (err != hipSuccess) {
    fprintf(stderr, "Failed to free device vector C (error code %s)!\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }

  // Free host memory
  free(h_A);
  free(h_B);
  free(h_C);

  //printf("Done\n");
  return 0;


}